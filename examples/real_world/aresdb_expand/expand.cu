
#include <hip/hip_runtime.h>
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// https://github.com/uber/aresdb/blob/a8d2aedc6850b10a6cc9381ba780800290b2756d/query/sort_reduce.cu#L252
// to 314

// this is a simplified version of the expand
// function that assumes all dim widths are 1
template <typename BaseCountsArray,
          typename IndexArray,
          typename InputKeysArray>
auto expand_parrot(const InputKeysArray &input_keys,
                   const BaseCountsArray &base_counts,
                   const IndexArray &indices,
                   int capacity) {
    auto counts = base_counts.deltas()
                    .gather(indices)
                    .sums()
                    .min(capacity)
                    .prepend(0)
                    .deltas()
                    .cycle({input_keys.size()});
    return input_keys.replicate(counts);
}
