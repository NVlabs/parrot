#include "hip/hip_runtime.h"
#include "parrot.hpp"

auto softmax(auto matrix) {
    auto cols = matrix.shape()[1];
    auto z    = matrix - matrix.template maxr<2>().replicate(cols);
    auto num  = z.exp();
    auto den  = num.template sum<2>();
    return num / den.replicate(cols);
}

int main() {
    auto matrix = parrot::range(6).as<float>().reshape({2, 3});
    softmax(matrix).print();
}
