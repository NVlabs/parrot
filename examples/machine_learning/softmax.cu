#include "hip/hip_runtime.h"
#include "parrot.hpp"

using namespace parrot::literals;

auto softmax(auto matrix) {
    auto cols = matrix.shape()[1];
    auto z    = matrix - matrix.maxr(2_ic).replicate(cols);
    auto num  = z.exp();
    auto den  = num.sum(2_ic);
    return num / den.replicate(cols);
}

int main() {
    auto matrix = parrot::range(6).as<float>().reshape({2, 3});
    softmax(matrix).print();
}
