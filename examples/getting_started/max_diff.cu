#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parrot.hpp"

int main() {
    // Find maximum absolute difference between consecutive elements
    auto data     = parrot::array({10, 5, 8, 3, 12, 1});
    auto max_diff = data  //
                      .deltas()
                      .abs()
                      .maxr()
                      .print();  // 11 (12 - 1)
}
