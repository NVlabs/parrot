#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>
#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Test deltas function
TEST_CASE("ParrotTest - DeltasTest") {
    auto arr    = parrot::array({1, 3, 6, 10});
    auto result = arr.deltas().sum();
    CHECK_EQ(result.value(), 9);  // sum of 2,3,4
}

// Test maxr function
TEST_CASE("ParrotTest - MaxrTest") {
    auto arr    = parrot::array({1, 5, 3, 2});
    auto result = arr.maxr();
    CHECK_EQ(result.value(), 5);
}

// Test minr function
TEST_CASE("ParrotTest - MinrTest") {
    auto arr    = parrot::array({1, 5, 3, 2});
    auto result = arr.minr();
    CHECK_EQ(result.value(), 1);
}

// Test minr with empty array (using only initial value)
TEST_CASE("ParrotTest - MinrEmptyTest") {
    auto arr    = parrot::array<int>({});
    auto result = arr.minr();
    CHECK_EQ(result.value(), std::numeric_limits<int>::max());
}

// Test minr with negative values
TEST_CASE("ParrotTest - MinrNegativeTest") {
    auto arr    = parrot::array({-1, -5, 3, 2});
    auto result = arr.minr();
    CHECK_EQ(result.value(), -5);
}

// Test minmax function
TEST_CASE("ParrotTest - MinmaxTest") {
    auto arr    = parrot::array({3, 1, 7, 5, 2});
    auto result = arr.minmax().to_host();
    REQUIRE_EQ(result.size(), 1);
    CHECK_EQ(result[0].first, 1);   // minimum value
    CHECK_EQ(result[0].second, 7);  // maximum value
}

// Test minmax with negative values
TEST_CASE("ParrotTest - MinmaxNegativeTest") {
    auto arr    = parrot::array({-3, 1, -7, 5, 2});
    auto result = arr.minmax().to_host();
    REQUIRE_EQ(result.size(), 1);
    CHECK_EQ(result[0].first, -7);  // minimum value
    CHECK_EQ(result[0].second, 5);  // maximum value
}

// Test any() method with all zeros
TEST_CASE("ParrotTest - AnyAllZerosTest") {
    auto arr    = parrot::array({0, 0, 0, 0});
    auto result = arr.any();
    CHECK_FALSE(result.value());
}

// Test any() method with some non-zeros
TEST_CASE("ParrotTest - AnySomeNonZerosTest") {
    auto arr    = parrot::array({0, 0, 3, 0});
    auto result = arr.any();
    CHECK(result.value());
}

// Test all() method with all non-zeros
TEST_CASE("ParrotTest - AllNonZerosTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.all();
    CHECK(result.value());
}

// Test all() method with some zeros
TEST_CASE("ParrotTest - AllSomeZerosTest") {
    auto arr    = parrot::array({1, 0, 3, 4});
    auto result = arr.all();
    CHECK_FALSE(result.value());
}

// Test any() and all() with empty array
TEST_CASE("ParrotTest - AnyAllEmptyTest") {
    auto arr = parrot::array<int>({});
    CHECK_FALSE(
      arr.any().value());  // Empty array should return false for any()
    CHECK(
      arr.all()
        .value());  // Empty array should return true for all() (vacuously true)
}

// Test prod function
TEST_CASE("ParrotTest - ProdTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.prod();
    CHECK_EQ(result.value(), 24);  // product of 1,2,3,4 is 24
}

// Test prod with an empty array
TEST_CASE("ParrotTest - ProdEmptyTest") {
    auto arr    = parrot::array<int>({});
    auto result = arr.prod();
    CHECK_EQ(result.value(),
             1);  // product of an empty array is the identity (1)
}

// Test prod with floating point values
TEST_CASE("ParrotTest - ProdFloatTest") {
    auto arr    = parrot::array<float>({1.5F, 2.0F, 2.5F});
    auto result = arr.prod();
    CHECK(result.value() ==
          doctest::Approx(7.5F));  // product of 1.5*2.0*2.5 = 7.5
}

// Test reduce function with plus operation
TEST_CASE("ParrotTest - ReducePlusTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.reduce(0, parrot::add{});
    CHECK_EQ(result.value(), 10);
    CHECK_EQ(result.value(),
             arr.sum().value());  // Verify it matches the sum() function
}

// Test reduce function with multiplies operation
TEST_CASE("ParrotTest - ReduceMultipliesTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.reduce(1, parrot::mul{});
    CHECK_EQ(result.value(), 24);
    CHECK_EQ(result.value(),
             arr.prod().value());  // Verify it matches the prod() function
}

// Test reduce function with maximum operation
TEST_CASE("ParrotTest - ReduceMaximumTest") {
    auto arr    = parrot::array({1, 5, 3, 4});
    auto result = arr.reduce(std::numeric_limits<int>::lowest(), parrot::max{});
    CHECK_EQ(result.value(), 5);
    CHECK_EQ(result.value(),
             arr.maxr().value());  // Verify it matches the maxr() function
}

// Test reduce function with minimum operation
TEST_CASE("ParrotTest - ReduceMinimumTest") {
    auto arr    = parrot::array({5, 2, 3, 4});
    auto result = arr.reduce(std::numeric_limits<int>::max(), parrot::min{});
    CHECK_EQ(result.value(), 2);
    CHECK_EQ(result.value(),
             arr.minr().value());  // Verify it matches the minr() function
}

// Test stats::mode function
TEST_CASE("ParrotTest - StatsModeTest") {
    auto arr    = parrot::array({3, 1, 3, 1, 2, 3});
    auto result = parrot::stats::mode(arr);
    CHECK_EQ(result.value(), 3);  // 3 appears most frequently (3 times)
}

// Test stats::mode with single mode
TEST_CASE("ParrotTest - StatsModeSingleTest") {
    auto arr    = parrot::array({1, 2, 2, 3});
    auto result = parrot::stats::mode(arr);
    CHECK_EQ(result.value(), 2);  // 2 appears most frequently (2 times)
}

// Test stats::mode with all unique elements
TEST_CASE("ParrotTest - StatsModeUniqueTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = parrot::stats::mode(arr);
    CHECK_EQ(result.value(), 1);  // All elements appear once, returns smallest
}

// Test stats::mode with single element
TEST_CASE("ParrotTest - StatsModeSingleElementTest") {
    auto arr    = parrot::array({42});
    auto result = parrot::stats::mode(arr);
    CHECK_EQ(result.value(), 42);  // Single element is the mode
}

// Test stats::mode with negative numbers
TEST_CASE("ParrotTest - StatsModeNegativeTest") {
    auto arr    = parrot::array({-1, -2, -1, -3, -1});
    auto result = parrot::stats::mode(arr);
    CHECK_EQ(result.value(), -1);  // -1 appears most frequently (3 times)
}