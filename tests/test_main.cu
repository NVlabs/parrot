#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Include all test files
#include "test_advanced_operations.cu"
#include "test_array_operations.cu"
#include "test_basic_operations.cu"
#include "test_integration.cu"
#include "test_math_operations.cu"
#include "test_multidimensional.cu"
#include "test_reductions.cu"
#include "test_scans.cu"
#include "test_sorting.cu"
#include "test_top10.cu"

// Additional integration tests can go here
TEST_CASE("ParrotTest - ChainedExampleTest") {
    auto arr    = parrot::array({3, 6, 9, 1});
    auto result = arr.append(1).sort().deltas().maxr();
    CHECK_EQ(result.value(),
             3);  // max delta in sorted [1,1,3,6,9] is 3 (between 6 and 9)
}

// Test the Python example min_differ_sum_double
TEST_CASE("ParrotTest - MinDifferSumDoubleTest") {
    auto result = parrot::range(20).min(5).differ().sum().dble();
    CHECK_EQ(result.value(), 8);
}

// Test replicate function with basic array
TEST_CASE("ParrotTest - ReplicateBasicTest") {
    auto arr      = parrot::array({1, 2, 3});
    auto result   = arr.replicate(2);
    auto expected = parrot::array({1, 1, 2, 2, 3, 3});
    CHECK(check_match(result, expected));
}

// Test replicate function with n=1 (should be identity)
TEST_CASE("ParrotTest - ReplicateIdentityTest") {
    auto arr      = parrot::array({5, 10, 15});
    auto result   = arr.replicate(1);
    auto expected = parrot::array({5, 10, 15});
    CHECK(check_match(result, expected));
}

// Test replicate function with larger n
TEST_CASE("ParrotTest - ReplicateLargeNTest") {
    auto arr      = parrot::array({7, 8});
    auto result   = arr.replicate(3);
    auto expected = parrot::array({7, 7, 7, 8, 8, 8});
    CHECK(check_match(result, expected));
}

// Test replicate function with single element
TEST_CASE("ParrotTest - ReplicateSingleElementTest") {
    auto arr      = parrot::array({42});
    auto result   = arr.replicate(4);
    auto expected = parrot::array({42, 42, 42, 42});
    CHECK(check_match(result, expected));
}

// Test replicate function with invalid n
TEST_CASE("ParrotTest - ReplicateInvalidNTest") {
    auto arr = parrot::array({1, 2, 3});
    CHECK_THROWS_AS((void)arr.replicate(0), std::invalid_argument);
    CHECK_THROWS_AS((void)arr.replicate(-1), std::invalid_argument);
}

// Test cross function with basic arrays
TEST_CASE("ParrotTest - CrossBasicTest") {
    auto arr1   = parrot::array({1, 2});
    auto arr2   = parrot::array({'a', 'b'});
    auto result = arr1.cross(arr2);

    CHECK_EQ(result.size(), 4);
    auto host_result = result.to_host();

    // Check the cartesian product: [(1, a), (1, b), (2, a), (2, b)]
    CHECK_EQ(host_result[0].first, 1);
    CHECK_EQ(host_result[0].second, 'a');
    CHECK_EQ(host_result[1].first, 1);
    CHECK_EQ(host_result[1].second, 'b');
    CHECK_EQ(host_result[2].first, 2);
    CHECK_EQ(host_result[2].second, 'a');
    CHECK_EQ(host_result[3].first, 2);
    CHECK_EQ(host_result[3].second, 'b');
}

// Test cross function with different sized arrays
TEST_CASE("ParrotTest - CrossDifferentSizesTest") {
    auto arr1   = parrot::array({10, 20, 30});
    auto arr2   = parrot::array({1, 2});
    auto result = arr1.cross(arr2);

    CHECK_EQ(result.size(), 6);
    auto host_result = result.to_host();

    // Check the cartesian product: [(10, 1), (10, 2), (20, 1), (20, 2), (30,
    // 1), (30, 2)]
    CHECK_EQ(host_result[0].first, 10);
    CHECK_EQ(host_result[0].second, 1);
    CHECK_EQ(host_result[1].first, 10);
    CHECK_EQ(host_result[1].second, 2);
    CHECK_EQ(host_result[2].first, 20);
    CHECK_EQ(host_result[2].second, 1);
    CHECK_EQ(host_result[3].first, 20);
    CHECK_EQ(host_result[3].second, 2);
    CHECK_EQ(host_result[4].first, 30);
    CHECK_EQ(host_result[4].second, 1);
    CHECK_EQ(host_result[5].first, 30);
    CHECK_EQ(host_result[5].second, 2);
}

// Test cross function with single element arrays
TEST_CASE("ParrotTest - CrossSingleElementTest") {
    auto arr1   = parrot::array({5});
    auto arr2   = parrot::array({100});
    auto result = arr1.cross(arr2);

    CHECK_EQ(result.size(), 1);
    auto host_result = result.to_host();

    CHECK_EQ(host_result[0].first, 5);
    CHECK_EQ(host_result[0].second, 100);
}

// Test cross function with empty arrays
TEST_CASE("ParrotTest - CrossEmptyArrayTest") {
    auto arr1 = parrot::array<int>({});
    auto arr2 = parrot::array({1, 2, 3});
    CHECK_THROWS_AS(arr1.cross(arr2), std::invalid_argument);

    auto arr3 = parrot::array({1, 2, 3});
    auto arr4 = parrot::array<int>({});
    CHECK_THROWS_AS(arr3.cross(arr4), std::invalid_argument);
}

// Test cross function with floating point arrays
TEST_CASE("ParrotTest - CrossFloatTest") {
    auto arr1   = parrot::array<float>({1.5F, 2.5F});
    auto arr2   = parrot::array<float>({10.0F, 20.0F});
    auto result = arr1.cross(arr2);

    CHECK_EQ(result.size(), 4);
    auto host_result = result.to_host();

    CHECK_EQ(host_result[0].first, 1.5F);
    CHECK_EQ(host_result[0].second, 10.0F);
    CHECK_EQ(host_result[1].first, 1.5F);
    CHECK_EQ(host_result[1].second, 20.0F);
    CHECK_EQ(host_result[2].first, 2.5F);
    CHECK_EQ(host_result[2].second, 10.0F);
    CHECK_EQ(host_result[3].first, 2.5F);
    CHECK_EQ(host_result[3].second, 20.0F);
}

// Test combined usage of replicate and cross
TEST_CASE("ParrotTest - ReplicateAndCrossCombinedTest") {
    auto arr1 = parrot::array({1, 2});
    auto arr2 = parrot::array({10, 20});

    // Test replicate followed by operations
    auto replicated          = arr1.replicate(2);
    auto expected_replicated = parrot::array({1, 1, 2, 2});
    CHECK(check_match(replicated, expected_replicated));

    // Test cross product
    auto crossed = arr1.cross(arr2);
    CHECK_EQ(crossed.size(), 4);

    // Test that replicate and cross work together as expected
    auto cross_result = arr1.cross(arr2);
    auto host_cross   = cross_result.to_host();

    // Verify the cross product is correct
    CHECK_EQ(host_cross[0].first, 1);
    CHECK_EQ(host_cross[0].second, 10);
    CHECK_EQ(host_cross[1].first, 1);
    CHECK_EQ(host_cross[1].second, 20);
    CHECK_EQ(host_cross[2].first, 2);
    CHECK_EQ(host_cross[2].second, 10);
    CHECK_EQ(host_cross[3].first, 2);
    CHECK_EQ(host_cross[3].second, 20);
}