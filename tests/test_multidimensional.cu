#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <sstream>
#include <stdexcept>
#include <string>
#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Test the stats::norm_cdf function
TEST_CASE("ParrotTest - StatsFunctions") {
    SUBCASE("norm_cdf function") {
        auto arr    = parrot::array<float>({0.0F, 1.0F, -1.0F});
        auto result = parrot::stats::norm_cdf(arr);

        // Expected values from standard normal CDF:
        // norm_cdf(0) = 0.5
        // norm_cdf(1) ≈ 0.8413
        // norm_cdf(-1) ≈ 0.1587
        auto expected = parrot::array<float>({0.5F, 0.8413447F, 0.1586553F});

        CHECK(result.size() == 3);
        auto result_host = result.to_host();

        CHECK(doctest::Approx(result_host[0]) == expected.to_host()[0]);
        CHECK(doctest::Approx(result_host[1]).epsilon(0.0001) ==
              expected.to_host()[1]);
        CHECK(doctest::Approx(result_host[2]).epsilon(0.0001) ==
              expected.to_host()[2]);
    }
}

// Test keep function with a range and stencil as in the example
TEST_CASE("ParrotTest - KeepRangeStencilTest") {
    auto stencil = parrot::array({0, 1, 1, 0, 0, 1, 0, 1});
    // parrot::range(8) -> [1, 2, 3, 4, 5, 6, 7, 8]
    // .minus(1) -> [0, 1, 2, 3, 4, 5, 6, 7]
    // .keep(stencil) -> keep elements where stencil is 1 -> indices [1, 2, 5,
    // 7]
    auto indices  = parrot::range(8).minus(1).keep(stencil);
    auto expected = parrot::array({1, 2, 5, 7});
    CHECK(check_match(indices, expected));
}

// Test the shape functionality
TEST_CASE("ParrotTest - ShapeTest") {
    // Test shape of range
    auto arr1 = parrot::range(10);
    REQUIRE_EQ(arr1.shape().size(), 1);
    CHECK_EQ(arr1.shape()[0], 10);

    // Test shape after operations
    auto arr2 = arr1.times(2);
    REQUIRE_EQ(arr2.shape().size(), 1);
    CHECK_EQ(arr2.shape()[0], 10);

    // Test shape after append
    auto arr3 = arr1.append(11);
    REQUIRE_EQ(arr3.shape().size(), 1);
    CHECK_EQ(arr3.shape()[0], 11);

    // Test shape after prepend
    auto arr4 = arr1.prepend(0);
    REQUIRE_EQ(arr4.shape().size(), 1);
    CHECK_EQ(arr4.shape()[0], 11);

    // Test shape after take
    auto arr5 = arr1.take(5);
    REQUIRE_EQ(arr5.shape().size(), 1);
    CHECK_EQ(arr5.shape()[0], 5);

    // Test shape after filtering
    auto stencil = parrot::array({1, 0, 1, 0, 1, 0, 1, 0, 1, 0});
    auto arr6    = arr1.keep(stencil);
    REQUIRE_EQ(arr6.shape().size(), 1);
    CHECK_EQ(arr6.shape()[0], 5);  // 5 elements kept
}

// Test reshape function (same size)
TEST_CASE("ParrotTest - ReshapeTest") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6});

    // Reshape to 2x3 (same total size)
    auto reshaped = arr.reshape({2, 3});

    // Check that the shape is correct
    auto shape = reshaped.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);
    CHECK_EQ(shape[1], 3);

    // Verify the total size remains the same
    CHECK_EQ(reshaped.size(), 6);

    // Original array should be unchanged
    REQUIRE_EQ(arr.shape().size(), 1);
    CHECK_EQ(arr.shape()[0], 6);
    CHECK_EQ(arr.size(), 6);

    // Check content
    auto expected = parrot::array({1, 2, 3, 4, 5, 6}).reshape({2, 3});
    CHECK(check_match(reshaped, expected));
}

// Test reshape function with truncation (smaller total size)
TEST_CASE("ParrotTest - ReshapeTruncateTest") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6});

    // Reshape to 2x2 (smaller total size, truncates to {1, 2, 3, 4})
    auto reshaped = arr.reshape({2, 2});

    // Check that the shape is correct
    auto shape = reshaped.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);
    CHECK_EQ(shape[1], 2);

    // Verify the total size is updated
    CHECK_EQ(reshaped.size(), 4);

    // Original array should be unchanged
    REQUIRE_EQ(arr.shape().size(), 1);
    CHECK_EQ(arr.shape()[0], 6);
    CHECK_EQ(arr.size(), 6);

    // Check content
    auto expected = parrot::array({1, 2, 3, 4});
    CHECK(check_match(reshaped, expected));

    // Check print output
    std::stringstream ss{};
    reshaped.print(ss);
    std::string const expected_print = "1 2\n3 4\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test reshape function with larger shape (should throw exception)
TEST_CASE("ParrotTest - ReshapeLargerSizeTest") {
    auto arr = parrot::array({1, 2, 3});
    // Reshape to 2x3 (larger total size 6 > 3)
    CHECK_THROWS_AS(static_cast<void>(arr.reshape({2, 3})),
                    std::invalid_argument);
}

// Test cycle function with data cycling (larger total size)
TEST_CASE("ParrotTest - CycleLargerSizeTest") {
    auto arr = parrot::array({1, 2, 3});

    // Cycle to 2x3 (larger total size 6 > 3, cycles data to {1, 2, 3, 1, 2, 3})
    auto cycled = arr.cycle({2, 3});

    // Check that the shape is correct
    auto shape = cycled.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);
    CHECK_EQ(shape[1], 3);

    // Verify the total size is updated
    CHECK_EQ(cycled.size(), 6);

    // Original array should be unchanged
    REQUIRE_EQ(arr.shape().size(), 1);
    CHECK_EQ(arr.shape()[0], 3);
    CHECK_EQ(arr.size(), 3);

    // Check content
    auto expected = parrot::array({1, 2, 3, 1, 2, 3});
    CHECK(check_match(cycled, expected));

    // Check print output
    std::stringstream ss{};
    cycled.print(ss);
    std::string const expected_print = "1 2 3\n1 2 3\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test cycle with equal size (behaves like reshape)
TEST_CASE("ParrotTest - CycleEqualSizeTest") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6});

    // Cycle to 2x3 (same total size as original)
    auto cycled = arr.cycle({2, 3});

    // Check that the shape is correct
    auto shape = cycled.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);
    CHECK_EQ(shape[1], 3);

    // Verify the total size remains the same
    CHECK_EQ(cycled.size(), 6);

    // Original array should be unchanged
    REQUIRE_EQ(arr.shape().size(), 1);
    CHECK_EQ(arr.shape()[0], 6);
    CHECK_EQ(arr.size(), 6);

    // Check content
    CHECK(check_match(cycled, arr));  // Should contain the same data

    // Check print output
    std::stringstream ss{};
    cycled.print(ss);
    std::string const expected_print = "1 2 3\n4 5 6\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test cycle with smaller size (truncates like reshape)
TEST_CASE("ParrotTest - CycleSmallerSizeTest") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6});

    // Cycle to 2x2 (smaller total size 4 < 6)
    auto cycled = arr.cycle({2, 2});

    // Check that the shape is correct
    auto shape = cycled.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);
    CHECK_EQ(shape[1], 2);

    // Verify the total size is updated
    CHECK_EQ(cycled.size(), 4);

    // Check content
    auto expected = parrot::array({1, 2, 3, 4});
    CHECK(check_match(cycled, expected));

    // Check print output
    std::stringstream ss{};
    cycled.print(ss);
    std::string const expected_print = "1 2\n3 4\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test multidimensional printing
TEST_CASE("ParrotTest - MultidimensionalPrintTest") {
    auto arr      = parrot::array({1, 2, 3, 4, 5, 6});
    auto reshaped = arr.reshape({2, 3});

    std::stringstream ss{};
    reshaped.print(ss);
    std::string const expected = "1 2 3\n4 5 6\n";
    CHECK_EQ(ss.str(), expected);

    // Test 3D printing
    auto arr3d_src = parrot::array({1, 2, 3, 4, 5, 6, 7, 8});
    auto arr3d     = arr3d_src.reshape({2, 2, 2});
    std::stringstream ss3d{};
    arr3d.print(ss3d);
    // Expected: Layer 0
    // 1 2 3 4 Layer 1 5 6 7 8(or similar)
    // Just check it doesn't crash and produces output for now
    CHECK(!ss3d.str().empty());
}

// Test rank method
TEST_CASE("ParrotTest - RankTest") {
    // Test rank of a 1D array
    auto arr1 = parrot::array({1, 2, 3, 4});
    CHECK_EQ(arr1.rank(), 1);

    // Test rank of a 2D array
    auto arr2 = arr1.reshape({2, 2});
    CHECK_EQ(arr2.rank(), 2);

    // Test rank of a 3D array
    auto arr3  = parrot::array({1, 2, 3, 4, 5, 6, 7, 8});
    auto arr3d = arr3.reshape({2, 2, 2});
    CHECK_EQ(arr3d.rank(), 3);

    // Test rank of a scalar
    auto scalar = parrot::scalar(42);
    CHECK_EQ(scalar.rank(), 0);
}

// Test scalar constructor
TEST_CASE("ParrotTest - ScalarConstructorTest") {
    // Create a scalar array with value 42
    auto scalar = parrot::scalar(42);
    CHECK_EQ(scalar.rank(), 0);
    CHECK_EQ(scalar.value(), 42);
    CHECK_EQ(scalar.size(), 1);  // Size is 1 for scalar

    // Test the factory function for scalars
    auto scalar2 = parrot::scalar(99);
    CHECK_EQ(scalar2.rank(), 0);
    CHECK_EQ(scalar2.value(), 99);
    CHECK_EQ(scalar2.size(), 1);
}

// Test repeat method
TEST_CASE("ParrotTest - RepeatTest") {
    auto scalar = parrot::scalar(7);

    // Repeat to create a 6-element array
    auto repeated = scalar.repeat(6);
    CHECK_EQ(repeated.size(), 6);

    // Check that the shape is correct (1D)
    auto shape = repeated.shape();
    REQUIRE_EQ(shape.size(), 1);
    CHECK_EQ(shape[0], 6);

    // Check content
    auto expected = parrot::array({7, 7, 7, 7, 7, 7});
    CHECK(check_match(repeated, expected));

    // Check print output
    std::stringstream ss{};
    repeated.print(ss);
    std::string const expected_print = "7 7 7 7 7 7\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test repeat method with invalid input
TEST_CASE("ParrotTest - RepeatInvalidTest") {
    // Create a non-scalar array
    auto arr = parrot::array({1, 2, 3});
    CHECK_THROWS_AS(static_cast<void>(arr.repeat(5)),
                    std::invalid_argument);  // Should throw exception

    // Create a scalar array
    auto scalar = parrot::scalar(7);
    CHECK_THROWS_AS(static_cast<void>(scalar.repeat(0)),
                    std::invalid_argument);  // Should throw for n=0
    CHECK_THROWS_AS(static_cast<void>(scalar.repeat(-1)),
                    std::invalid_argument);  // Should throw for n<0
}

// Test matrix function
TEST_CASE("ParrotTest - MatrixTest") {
    // Create a matrix with value 7 and shape {3, 4}
    auto mat = parrot::matrix(7, {3, 4});
    CHECK_EQ(mat.size(), 12);

    // Check that the shape is correct (2D)
    auto shape = mat.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 3);
    CHECK_EQ(shape[1], 4);

    // Check content - all elements should be 7
    auto host_vals = mat.to_host();
    REQUIRE_EQ(host_vals.size(), 12);
    for (int const val : host_vals) { CHECK_EQ(val, 7); }

    // Check print output
    std::stringstream ss{};
    mat.print(ss);
    std::string const expected_print = "7 7 7 7\n7 7 7 7\n7 7 7 7\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test matrix function with invalid inputs
TEST_CASE("ParrotTest - MatrixInvalidTest") {
    CHECK_THROWS_AS(parrot::matrix(5, {10}),
                    std::invalid_argument);  // Shape must have > 1 dimension
    CHECK_THROWS_AS(parrot::matrix(5, {2, 3, 4}),
                    std::invalid_argument);  // Shape must have <= 2 dimensions
    CHECK_THROWS_AS(parrot::matrix(5, {}),
                    std::invalid_argument);  // Shape cannot be empty
}

// Test nested initializer list matrix function
TEST_CASE("ParrotTest - NestedMatrixTest") {
    // Create a 2x3 matrix with integers
    auto mat = parrot::matrix({{1, 2, 3}, {4, 5, 6}});

    // Check size and shape
    CHECK_EQ(mat.size(), 6);
    auto shape = mat.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 2);  // rows
    CHECK_EQ(shape[1], 3);  // cols

    // Check content (row-major order)
    auto expected = parrot::array({1, 2, 3, 4, 5, 6});
    CHECK(check_match(mat, expected));

    // Check print output
    std::stringstream ss{};
    mat.print(ss);
    std::string const expected_print = "1 2 3\n4 5 6\n";
    CHECK_EQ(ss.str(), expected_print);
}

// Test nested initializer list matrix with doubles
TEST_CASE("ParrotTest - NestedMatrixDoubleTest") {
    // Create a 3x2 matrix with doubles
    auto mat = parrot::matrix({{1.5, 2.5}, {3.5, 4.5}, {5.5, 6.5}});

    // Check size and shape
    CHECK_EQ(mat.size(), 6);
    auto shape = mat.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 3);  // rows
    CHECK_EQ(shape[1], 2);  // cols

    // Check content
    auto expected = parrot::array({1.5, 2.5, 3.5, 4.5, 5.5, 6.5});
    CHECK(check_match(mat, expected));
}

// Test nested initializer list matrix with single element
TEST_CASE("ParrotTest - NestedMatrixSingleElementTest") {
    // Create a 1x1 matrix
    auto mat = parrot::matrix({{42}});

    // Check size and shape
    CHECK_EQ(mat.size(), 1);
    auto shape = mat.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 1);  // rows
    CHECK_EQ(shape[1], 1);  // cols

    // Check content
    auto expected = parrot::array({42});
    CHECK(check_match(mat, expected));
}

// Test nested initializer list matrix with invalid inputs
TEST_CASE("ParrotTest - NestedMatrixInvalidTest") {
    // Empty nested list - need explicit type since compiler can't deduce from
    // empty list
    std::initializer_list<std::initializer_list<int>> empty_nested{};
    CHECK_THROWS_AS(parrot::matrix(empty_nested), std::invalid_argument);

    // Empty inner list - need explicit type since compiler can't deduce from
    // empty inner list
    std::initializer_list<std::initializer_list<int>> empty_inner{{}};
    CHECK_THROWS_AS(parrot::matrix(empty_inner), std::invalid_argument);

    // Mismatched row lengths
    CHECK_THROWS_AS(parrot::matrix({{1, 2, 3}, {4, 5}}), std::invalid_argument);

    // Another mismatched case
    CHECK_THROWS_AS(parrot::matrix({{1, 2}, {3, 4, 5}, {6, 7}}),
                    std::invalid_argument);
}

// Test transpose function for a 2x3 matrix
TEST_CASE("ParrotTest - Transpose2x3Test") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6})
                 .reshape({2, 3});  // [[1,2,3],[4,5,6]]
    auto transposed = arr.transpose();
    auto expected   = parrot::array({1, 4, 2, 5, 3, 6})
                      .reshape({3, 2});  // [[1,4],[2,5],[3,6]]

    // Check shape
    auto shape = transposed.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 3);
    CHECK_EQ(shape[1], 2);
    CHECK_EQ(transposed.size(), 6);

    // Check content
    CHECK(check_match(transposed, expected));
}

// Test transpose function for a 3x2 matrix (double transpose)
TEST_CASE("ParrotTest - Transpose3x2Test") {
    auto arr = parrot::array({1, 4, 2, 5, 3, 6})
                 .reshape({3, 2});            // [[1,4],[2,5],[3,6]]
    auto transposed_once  = arr.transpose();  // Should be [[1,2,3],[4,5,6]]
    auto transposed_twice = transposed_once
                              .transpose();  // Should be back to original

    // Check shape of single transpose
    auto shape_once = transposed_once.shape();
    REQUIRE_EQ(shape_once.size(), 2);
    CHECK_EQ(shape_once[0], 2);
    CHECK_EQ(shape_once[1], 3);
    CHECK_EQ(transposed_once.size(), 6);

    // Check shape of double transpose
    auto shape_twice = transposed_twice.shape();
    REQUIRE_EQ(shape_twice.size(), 2);
    CHECK_EQ(shape_twice[0], 3);
    CHECK_EQ(shape_twice[1], 2);
    CHECK_EQ(transposed_twice.size(), 6);

    // Check content of double transpose (should match original)
    CHECK(check_match(transposed_twice, arr));
}

// Test transpose function for a single row matrix (1xN)
TEST_CASE("ParrotTest - TransposeSingleRowTest") {
    auto arr = parrot::array({1, 2, 3, 4}).reshape({1, 4});  // [[1,2,3,4]]
    auto transposed = arr.transpose();
    auto expected   = parrot::array({1, 2, 3, 4})
                      .reshape({4, 1});  // [[1],[2],[3],[4]]

    // Check shape
    auto shape = transposed.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 4);
    CHECK_EQ(shape[1], 1);
    CHECK_EQ(transposed.size(), 4);

    // Check content
    CHECK(check_match(transposed, expected));
}

// Test transpose function for a single column matrix (Nx1)
TEST_CASE("ParrotTest - TransposeSingleColumnTest") {
    auto arr = parrot::array({1, 2, 3, 4})
                 .reshape({4, 1});  // [[1],[2],[3],[4]]
    auto transposed = arr.transpose();
    auto expected = parrot::array({1, 2, 3, 4}).reshape({1, 4});  // [[1,2,3,4]]

    // Check shape
    auto shape = transposed.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 1);
    CHECK_EQ(shape[1], 4);
    CHECK_EQ(transposed.size(), 4);

    // Check content
    CHECK(check_match(transposed, expected));
}

// Test transpose function for a 1x1 matrix
TEST_CASE("ParrotTest - Transpose1x1Test") {
    auto arr        = parrot::array({42}).reshape({1, 1});  // [[42]]
    auto transposed = arr.transpose();
    auto expected   = parrot::array({42}).reshape({1, 1});  // [[42]]

    // Check shape
    auto shape = transposed.shape();
    REQUIRE_EQ(shape.size(), 2);
    CHECK_EQ(shape[0], 1);
    CHECK_EQ(shape[1], 1);
    CHECK_EQ(transposed.size(), 1);

    // Check content
    CHECK(check_match(transposed, expected));
}

// Test transpose function with invalid input (1D array)
TEST_CASE("ParrotTest - TransposeInvalid1DTest") {
    auto arr = parrot::array({1, 2, 3, 4});  // 1D array
    CHECK_THROWS_AS(static_cast<void>(arr.transpose()), std::invalid_argument);
}

// Test transpose function with invalid input (3D array)
TEST_CASE("ParrotTest - TransposeInvalid3DTest") {
    auto arr = parrot::array({1, 2, 3, 4, 5, 6, 7, 8})
                 .reshape({2, 2, 2});  // 3D array
    CHECK_THROWS_AS(static_cast<void>(arr.transpose()), std::invalid_argument);
}

// Test scan with Axis=1 (column-wise)
TEST_CASE("ParrotTest - ScanColTest") {
    auto matrix = parrot::array({1, 2, 3, 4, 5, 6, 7, 8, 9}).reshape({3, 3});

    // Column-wise sums
    auto scan_col_sums     = matrix.scan<1>(parrot::add{});
    auto expected_col_sums = parrot::array({1,
                                            2,
                                            3,
                                            1 + 4,
                                            2 + 5,
                                            3 + 6,
                                            1 + 4 + 7,
                                            2 + 5 + 8,
                                            3 + 6 + 9})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_sums, expected_col_sums));

    // Column-wise prods
    auto scan_col_prods     = matrix.scan<1>(parrot::mul{});
    auto expected_col_prods = parrot::array({1,
                                             2,
                                             3,
                                             1 * 4,
                                             2 * 5,
                                             3 * 6,
                                             1 * 4 * 7,
                                             2 * 5 * 8,
                                             3 * 6 * 9})
                                .reshape({3, 3});
    CHECK(check_match(scan_col_prods, expected_col_prods));

    // Column-wise mins
    auto scan_col_mins     = matrix.scan<1>(parrot::min{});
    auto expected_col_mins = parrot::array({1, 2, 3, 1, 2, 3, 1, 2, 3})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_mins, expected_col_mins));

    // Column-wise maxs
    auto scan_col_maxs     = matrix.scan<1>(parrot::max{});
    auto expected_col_maxs = parrot::array({1, 2, 3, 4, 5, 6, 7, 8, 9})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_maxs, expected_col_maxs));
}