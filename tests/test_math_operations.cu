#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES.
 * All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cmath>
#include <cstddef>
#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Test double function
TEST_CASE("ParrotTest - DoubleTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.dble().sum();
    CHECK_EQ(result.value(), 20);  // sum of [2,4,6,8] = 20
}

// Test half function
TEST_CASE("ParrotTest - HalfTest") {
    auto arr    = parrot::array({2, 4, 6, 8});
    auto result = arr.half().sum();
    CHECK_EQ(result.value(), 10);  // sum of [1,2,3,4] = 10
}

// Test half with odd numbers
TEST_CASE("ParrotTest - HalfOddTest") {
    auto arr    = parrot::array({1, 3, 5, 7});
    auto result = arr.half().sum();
    CHECK_EQ(result.value(),
             6);  // sum of [0,1,2,3] = 6 (integer division truncates)
}

// Test half with floating point values
TEST_CASE("ParrotTest - HalfFloatTest") {
    auto arr    = parrot::array<float>({1.0F, 3.0F, 5.0F, 7.0F});
    auto result = arr.half().sum();
    CHECK(result.value() ==
          doctest::Approx(8.0F));  // sum of [0.5,1.5,2.5,3.5] = 8.0
}

// Test abs function with mixed sign values
TEST_CASE("ParrotTest - AbsTest") {
    auto arr    = parrot::array({-2, 3, -5, 7});
    auto result = arr.abs().sum();
    CHECK_EQ(result.value(), 17);  // sum of [2,3,5,7] = 17
}

// Test abs function with all negative values
TEST_CASE("ParrotTest - AbsNegativeTest") {
    auto arr    = parrot::array({-10, -20, -30});
    auto result = arr.abs().sum();
    CHECK_EQ(result.value(), 60);  // sum of [10,20,30] = 60
}

// Test abs function with floating point values
TEST_CASE("ParrotTest - AbsFloatTest") {
    auto arr    = parrot::array<float>({-1.5F, 2.5F, -3.5F});
    auto result = arr.abs().sum();
    CHECK(result.value() ==
          doctest::Approx(7.5F));  // sum of [1.5,2.5,3.5] = 7.5
}

// Test log function with positive values
TEST_CASE("ParrotTest - LogTest") {
    auto arr             = parrot::array<float>({1.0F, 2.0F, 3.0F});
    auto result          = arr.log().sum();
    float const expected = std::log(1.0F) + std::numbers::ln2_v<float> +
                           std::log(3.0F);
    CHECK(result.value() == doctest::Approx(expected));
}

// Test log function with e values
TEST_CASE("ParrotTest - LogETest") {
    auto arr = parrot::array<float>(
      {1.0F, std::numbers::e_v<float>, std::exp(2.0F)});
    auto result = arr.log().sum();
    CHECK(result.value() ==
          doctest::Approx(3.0F));  // sum of log(1)=0, log(e)=1, log(e²)=2 = 3
}

// Test exp function with values
TEST_CASE("ParrotTest - ExpTest") {
    auto arr             = parrot::array<float>({0.0F, 1.0F, 2.0F});
    auto result          = arr.exp().sum();
    float const expected = 1.0F + std::numbers::e_v<float> + std::exp(2.0F);
    CHECK(result.value() == doctest::Approx(expected));
}

// Test exp function with log values (they should cancel out)
TEST_CASE("ParrotTest - ExpLogTest") {
    auto arr    = parrot::array<float>({1.0F, 2.0F, 3.0F});
    auto result = arr.log().exp();
    CHECK(result.to_host()[0] == doctest::Approx(1.0F));
    CHECK(result.to_host()[1] == doctest::Approx(2.0F));
    CHECK(result.to_host()[2] == doctest::Approx(3.0F));
}

// Test sqrt function with perfect squares
TEST_CASE("ParrotTest - SqrtPerfectSquaresTest") {
    auto arr    = parrot::array<float>({4.0F, 9.0F, 16.0F, 25.0F});
    auto result = arr.sqrt().sum();
    CHECK(result.value() == doctest::Approx(14.0F));  // sum of [2,3,4,5] = 14
}

// Test sqrt function with non-perfect squares
TEST_CASE("ParrotTest - SqrtTest") {
    auto arr             = parrot::array<float>({2.0F, 3.0F, 5.0F});
    auto result          = arr.sqrt().sum();
    float const expected = std::numbers::sqrt2_v<float> +
                           std::numbers::sqrt3_v<float> + std::sqrt(5.0F);
    CHECK(result.value() == doctest::Approx(expected));
}

// Test sq function (square)
TEST_CASE("ParrotTest - SqTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.sq().sum();
    CHECK_EQ(result.value(), 30);  // sum of [1,4,9,16] = 30
}

// Test sq function with negative values
TEST_CASE("ParrotTest - SqNegativeTest") {
    auto arr    = parrot::array({-1, -2, 3});
    auto result = arr.sq().sum();
    CHECK_EQ(result.value(), 14);  // sum of [1,4,9] = 14
}

// Test odd function with mixed values
TEST_CASE("ParrotTest - OddTest") {
    auto arr    = parrot::array({1, 2, 3, 4, 5});
    auto result = arr.odd().sum();
    CHECK_EQ(result.value(), 3);  // 1,3,5 are odd, so sum of [1,0,1,0,1] = 3
}

// Test odd function with all even values
TEST_CASE("ParrotTest - OddAllEvenTest") {
    auto arr    = parrot::array({2, 4, 6, 8});
    auto result = arr.odd().sum();
    CHECK_EQ(result.value(), 0);  // all even, so sum of [0,0,0,0] = 0
}

// Test even function with mixed values
TEST_CASE("ParrotTest - EvenTest") {
    auto arr    = parrot::array({1, 2, 3, 4, 5});
    auto result = arr.even().sum();
    CHECK_EQ(result.value(), 2);  // 2,4 are even, so sum of [0,1,0,1,0] = 2
}

// Test even function with all odd values
TEST_CASE("ParrotTest - EvenAllOddTest") {
    auto arr    = parrot::array({1, 3, 5, 7});
    auto result = arr.even().sum();
    CHECK_EQ(result.value(), 0);  // all odd, so sum of [0,0,0,0] = 0
}

// Test rand with integer array
TEST_CASE("ParrotTest - RandIntTest") {
    auto arr         = parrot::array({10, 20, 30, 40});
    auto result      = arr.rand();
    auto result_host = result.to_host();
    auto arr_host    = arr.to_host();

    REQUIRE_EQ(result_host.size(), arr_host.size());
    for (size_t i = 0; i < result_host.size(); i++) {
        CHECK_GE(result_host[i], 0);
        // rand() generates in [0, N), so should be strictly less than N if N >
        // 0
        if (arr_host[i] > 0) {
            CHECK_LT(result_host[i], arr_host[i]);
        } else {
            CHECK_EQ(result_host[i], 0);  // rand(0) should be 0
        }
    }
}

// Test rand with floating point array
TEST_CASE("ParrotTest - RandFloatTest") {
    auto arr         = parrot::array<float>({10.0F, 20.0F, 0.0F, 40.0F});
    auto result      = arr.rand();
    auto result_host = result.to_host();
    auto arr_host    = arr.to_host();

    REQUIRE_EQ(result_host.size(), arr_host.size());
    for (size_t i = 0; i < result_host.size(); i++) {
        CHECK_GE(result_host[i], 0.0F);
        // randf() generates in [0, N), should be strictly less than N if N > 0
        if (arr_host[i] > 0.0F) {
            CHECK_LT(result_host[i], arr_host[i]);
        } else {
            CHECK_EQ(result_host[i], 0.0F);  // randf(0.0) should be 0.0
        }
    }
}