#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Test sums function (inclusive scan with addition)
TEST_CASE("ParrotTest - SumsTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.sums();
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({1, 3, 6, 10});
    CHECK(check_match(result, expected_arr));
}

// Test prods function (inclusive scan with multiplication)
TEST_CASE("ParrotTest - ProdsTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.prods();
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({1, 2, 6, 24});
    CHECK(check_match(result, expected_arr));
}

// Test mins function (inclusive scan with minimum)
TEST_CASE("ParrotTest - MinsTest") {
    auto arr    = parrot::array({3, 1, 4, 2});
    auto result = arr.mins();
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({3, 1, 1, 1});
    CHECK(check_match(result, expected_arr));
}

// Test maxs function (inclusive scan with maximum)
TEST_CASE("ParrotTest - MaxsTest") {
    auto arr    = parrot::array({3, 1, 4, 2});
    auto result = arr.maxs();
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({3, 3, 4, 4});
    CHECK(check_match(result, expected_arr));
}

// Test anys function (inclusive scan with logical OR)
TEST_CASE("ParrotTest - AnysTest") {
    auto arr    = parrot::array({0, 1, 0, 0, 2});  // Non-zero treated as true
    auto result = arr.anys();
    CHECK_EQ(result.size(), 5);
    auto expected_arr = parrot::array({0, 1, 1, 1, 1});  // Result is 0 or 1
    CHECK(check_match(result, expected_arr));
}

// Test alls function (inclusive scan with logical AND)
TEST_CASE("ParrotTest - AllsTest") {
    auto arr = parrot::array(
      {1, 1, 0, 1, 2});  // Zero treated as false, non-zero as true
    auto result = arr.alls();
    CHECK_EQ(result.size(), 5);
    auto expected_arr = parrot::array({1, 1, 0, 0, 0});  // Result is 0 or 1
    CHECK(check_match(result, expected_arr));
}

// Test scan function with plus operation
TEST_CASE("ParrotTest - ScanPlusTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.scan(parrot::add{});
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({1, 3, 6, 10});
    CHECK(check_match(result, expected_arr));
    CHECK(check_match(result,
                      arr.sums()));  // Verify it matches the sums() function
}

// Test scan function with multiplies operation
TEST_CASE("ParrotTest - ScanMultipliesTest") {
    auto arr    = parrot::array({1, 2, 3, 4});
    auto result = arr.scan(parrot::mul{});
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({1, 2, 6, 24});
    CHECK(check_match(result, expected_arr));
    CHECK(check_match(result,
                      arr.prods()));  // Verify it matches the prods() function
}

// Test scan function with minimum operation
TEST_CASE("ParrotTest - ScanMinimumTest") {
    auto arr    = parrot::array({3, 1, 4, 2});
    auto result = arr.scan(parrot::min{});
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({3, 1, 1, 1});
    CHECK(check_match(result, expected_arr));
    CHECK(check_match(result,
                      arr.mins()));  // Verify it matches the mins() function
}

// Test scan function with maximum operation
TEST_CASE("ParrotTest - ScanMaximumTest") {
    auto arr    = parrot::array({3, 1, 4, 2});
    auto result = arr.scan(parrot::max{});
    CHECK_EQ(result.size(), 4);
    auto expected_arr = parrot::array({3, 3, 4, 4});
    CHECK(check_match(result, expected_arr));
    CHECK(check_match(result,
                      arr.maxs()));  // Verify it matches the maxs() function
}

// Test scan with Axis=1 (column-wise)
TEST_CASE("ParrotTest - ScanColTest") {
    auto matrix = parrot::array({1, 2, 3, 4, 5, 6, 7, 8, 9}).reshape({3, 3});

    // Column-wise sums
    auto scan_col_sums     = matrix.scan<1>(parrot::add{});
    auto expected_col_sums = parrot::array({1,
                                         2,
                                         3,
                                         1 + 4,
                                         2 + 5,
                                         3 + 6,
                                         1 + 4 + 7,
                                         2 + 5 + 8,
                                         3 + 6 + 9})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_sums, expected_col_sums));

    // Column-wise prods
    auto scan_col_prods     = matrix.scan<1>(parrot::mul{});
    auto expected_col_prods = parrot::array({1,
                                          2,
                                          3,
                                          1 * 4,
                                          2 * 5,
                                          3 * 6,
                                          1 * 4 * 7,
                                          2 * 5 * 8,
                                          3 * 6 * 9})
                                .reshape({3, 3});
    CHECK(check_match(scan_col_prods, expected_col_prods));

    // Column-wise mins
    auto scan_col_mins     = matrix.scan<1>(parrot::min{});
    auto expected_col_mins = parrot::array({1, 2, 3, 1, 2, 3, 1, 2, 3})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_mins, expected_col_mins));

    // Column-wise maxs
    auto scan_col_maxs     = matrix.scan<1>(parrot::max{});
    auto expected_col_maxs = parrot::array({1, 2, 3, 4, 5, 6, 7, 8, 9})
                               .reshape({3, 3});
    CHECK(check_match(scan_col_maxs, expected_col_maxs));
}
