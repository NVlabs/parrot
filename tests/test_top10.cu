#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "parrot.hpp"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "test_common.hpp"

// Test the rain water test with rev
TEST_CASE("Top 10 - #1 Rain Water") {
    auto arr    = parrot::array({0, 1, 0, 2, 1, 0, 1, 3, 2, 1, 2, 1});
    auto result = arr.maxs().min(arr.rev().maxs().rev()).minus(arr).sum();
    CHECK_EQ(result.value(), 6);
}

// Test the MCO (Maximum Consecutive Ones) example
TEST_CASE("Top 10 - #2 Maximum Consecutive Ones (MCO)") {
    SUBCASE("Test case 1") {
        auto nums   = parrot::array({1, 1, 0, 1, 1, 1});
        auto result = nums.chunk_by_reduce(parrot::eq{}, parrot::add{}).maxr();
        CHECK_EQ(result.value(), 3);
    }

    SUBCASE("Test case 2") {
        auto nums   = parrot::array({1, 0, 1, 1, 0, 1});
        auto result = nums.chunk_by_reduce(parrot::eq{}, parrot::add{}).maxr();
        CHECK_EQ(result.value(), 2);
    }
}

// Test the LCIS (Longest Consecutive Increasing Subsequence) example
TEST_CASE("Top 10 - #3 LCIS") {
    SUBCASE("Test case 1") {
        auto nums   = parrot::array({1, 3, 5, 4, 7});
        auto result = nums.map_adj(parrot::lt{})
                        .chunk_by_reduce(parrot::eq{}, parrot::add{})
                        .maxr()
                        .add(1);
        CHECK_EQ(result.value(), 3);
    }

    SUBCASE("Test case 2") {
        auto nums   = parrot::array({2, 2, 2, 2, 2});
        auto result = nums.map_adj(parrot::lt{})
                        .chunk_by_reduce(parrot::eq{}, parrot::add{})
                        .maxr()
                        .add(1);
        CHECK_EQ(result.value(), 1);
    }
}

// Test the maximum gap example
TEST_CASE("Top 10 - #6 Maximum Gap") {
    SUBCASE("Test case 1") {
        auto nums   = parrot::array({3, 6, 9, 1});
        auto result = nums.append(nums.back()).sort().deltas().maxr();
        CHECK_EQ(result.value(), 3);
    }

    SUBCASE("Test case 2") {
        auto nums   = parrot::array({10});
        auto result = nums.append(nums.back()).sort().deltas().maxr();
        CHECK_EQ(result.value(), 0);
    }
}

// Test the maximum gap count example
TEST_CASE("Top 10 - #7 Maximum Gap Count") {
    SUBCASE("Test case 1") {
        auto nums   = parrot::array({3, 6, 9, 1});
        auto d      = nums.sort().deltas();
        auto result = (d.maxr() == d).sum();
        CHECK_EQ(result.value(), 2);
    }

    SUBCASE("Test case 2") {
        auto nums   = parrot::array({2, 5, 8, 1});
        auto d      = nums.sort().deltas();
        auto result = (d.maxr() == d).sum();
        CHECK_EQ(result.value(), 2);
    }

    SUBCASE("Test case 3") {
        auto nums   = parrot::array({10});
        auto d      = nums.sort().deltas();
        auto result = (d.maxr() == d).sum();
        CHECK_EQ(result.value(), 0);
    }
}

// Test the full sushi freshness example
TEST_CASE("Top 10 - #5 Sushi For Two") {
    SUBCASE("Test case 1") {
        auto sushi  = parrot::array({2, 2, 2, 1, 1, 2, 2});
        auto result = sushi.differ()
                        .where()
                        .prepend(0)
                        .append(sushi.size())
                        .deltas()
                        .map_adj(parrot::min{})
                        .dble()
                        .maxr();
        CHECK_EQ(result.value(), 4);
    }

    SUBCASE("Test case 2") {
        auto sushi  = parrot::array({1, 2, 1, 2, 1, 2});
        auto result = sushi.differ()
                        .where()
                        .prepend(0)
                        .append(sushi.size())
                        .deltas()
                        .map_adj(parrot::min{})
                        .dble()
                        .maxr();
        CHECK_EQ(result.value(), 2);
    }

    SUBCASE("Test case 3") {
        auto sushi  = parrot::array({2, 2, 1, 1, 1, 2, 2, 2, 2});
        auto result = sushi.differ()
                        .where()
                        .prepend(0)
                        .append(sushi.size())
                        .deltas()
                        .map_adj(parrot::min{})
                        .dble()
                        .maxr();
        CHECK_EQ(result.value(), 6);
    }
}

// Test the TCO (Three Consecutive Odds) example
TEST_CASE("Top 10 - #8 Three Consecutive Odds (TCO)") {
    SUBCASE("Test case 1") {
        auto arr    = parrot::array({2, 6, 4, 1});
        auto result = arr  //
                        .odd()
                        .chunk_by_reduce(parrot::eq{}, parrot::add{})
                        .maxr()
                        .gte(3);
        CHECK_EQ(result.value(), false);
    }

    SUBCASE("Test case 2") {
        auto arr    = parrot::array({1, 2, 34, 3, 4, 5, 7, 23, 12});
        auto result = arr  //
                        .odd()
                        .chunk_by_reduce(parrot::eq{}, parrot::add{})
                        .maxr()
                        .gte(3);
        CHECK_EQ(result.value(), true);
    }
}

// Test the skyline example
TEST_CASE("Top 10 - #9 Skyline") {
    auto heights = parrot::array({1, 0, 3, 2, 5, 4});
    auto result  = heights.maxs().uniq().size();
    auto result2 = heights.maxs().distinct().size();
    CHECK_EQ(result, 3);
    CHECK_EQ(result2, 3);
}

// Test the ocean view example
TEST_CASE("Top 10 - #10 Ocean View 3") {
    SUBCASE("Test case 1") {
        auto nums     = parrot::array({4, 2, 3, 1});
        auto result   = nums.rev().maxs().differ().prepend(1).rev().where();
        auto result2  = nums.rev().maxs().differ().rev().append(1).where();
        auto expected = parrot::array({1, 3, 4});
        check_match_eq(result, expected);
        check_match_eq(result2, expected);
    }

    SUBCASE("Test case 2") {
        auto nums     = parrot::array({4, 3, 2, 1});
        auto result   = nums.rev().maxs().differ().prepend(1).rev().where();
        auto result2  = nums.rev().maxs().differ().rev().append(1).where();
        auto expected = parrot::array({1, 2, 3, 4});
        check_match_eq(result, expected);
        check_match_eq(result2, expected);
    }

    SUBCASE("Test case 3") {
        auto nums     = parrot::array({1, 3, 2, 4});
        auto result   = nums.rev().maxs().differ().prepend(1).rev().where();
        auto result2  = nums.rev().maxs().differ().rev().append(1).where();
        auto expected = parrot::array({4});
        check_match_eq(result, expected);
        check_match_eq(result2, expected);
    }

    SUBCASE("Test case 4") {
        auto nums     = parrot::array({2, 2, 2, 2});
        auto result   = nums.rev().maxs().differ().prepend(1).rev().where();
        auto result2  = nums.rev().maxs().differ().rev().append(1).where();
        auto expected = parrot::array({4});
        check_match_eq(result, expected);
        check_match_eq(result2, expected);
    }
}
